#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <vector>
#include <algorithm>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

using namespace thrust;

int main( int argc, char* argv[] ) {
 
    // Size of vectors
    const int n = 1000000;
 
    // Input device vectors
    device_vector<double> a(n);
    device_vector<double> b(n);
    // Output device  vector
    device_vector<double> c(n);
 
    // Create counting iterators to describe our vectors
    auto begin_index = counting_iterator<int>(0);
    auto end_index   = begin_index + n;

    // Unary transform: a_i = sin^2(i/pi)
    transform(thrust::device,      // Execution policy
              begin_index,         // Input begin
              end_index,           // Input end
              a.begin(),           // Output begin
              [] __device__ (int i) {
      return sin(i/M_PI) * sin(i/M_PI);
    });

    // Unary transform: b_i = cos^2(i/pi)
    transform(thrust::device,
              begin_index,
              end_index,
              b.begin(),
             [] __device__ (int i) {
      return cos(i/M_PI) * cos(i/M_PI);
    });

    // Zip vectors a,b together
    auto begin_zip = make_zip_iterator(make_tuple(a.begin(), b.begin()));
    auto end_zip = begin_zip + n;

    // Binary transform: c_i = a_i + b_i
    transform(thrust::device,
              begin_zip,
              end_zip,
              c.begin(),
              [] __device__ (const tuple<double,double>& tuple_i) {
       double a_i, b_i;
       tie(a_i, b_i) = tuple_i;
       return a_i + b_i;   
    });

    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = reduce(thrust::device, c.begin(), c.end());
    sum /= n;
    if(sum - 1.0 < 0.0000001)
      printf("result correct\n");
    else
      printf("result incorrect!\n");

    return 0;
}
