#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <vector>
#include <algorithm>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/reduce.h>
#include <thrust/pair.h>

__device__ double sin_squared(double theta) {
  return sin(theta) * sin(theta);
} 

__device__ double cos_squared(double theta) {
  return cos(theta) * cos(theta);
}

int main( int argc, char* argv[] ) {
 
    // Size of vectors
    const int n = 1000000;
 
    // Input device vectors
    thrust::device_vector<double> a(n);
    thrust::device_vector<double> b(n);
    // Output device  vector
    thrust::device_vector<double> c(n);
 
    // Create counting iterators to describe our vectors
    const auto begin_index = thrust::counting_iterator<int>(0);
    const auto end_index   = thrust::counting_iterator<int>(n);

    // Unary transform: a_i = sin^2(i/pi)
    thrust::transform(begin_index,         // Input begin
                      end_index,           // Input end
                      a.begin(),           // Output begin
                      [] __device__ (int i) {
      return sin_squared(i/M_PI);
    });

    // Unary transform: b_i = cos^2(i/pi)
    thrust::transform(begin_index,         // Input begin
                      end_index,           // Input end
                      b.begin(),           // Output begin
                      [] __device__ (int i) {
      return cos_squared(i/M_PI);
    });

    // Tie our vectors a,b together
    auto begin_zip = thrust::make_zip_iterator(thrust::make_tuple(a.begin(), b.begin()));
    auto end_zip = thrust::make_zip_iterator(thrust::make_tuple(a.end(), b.end()));

    typedef thrust::tuple<double, double> double_tuple;

    // Binary transform: c_i = a_i + b_i
    thrust::transform(begin_zip,
                      end_zip,
                      c.begin(),
                      [] __device__ (const double_tuple& pair_i) {
       double a_i, b_i;
       thrust::tie(a_i, b_i) = pair_i;
       return a_i + b_i;   
    });

    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = thrust::reduce(c.begin(), c.end());
    sum /= n;
    if(sum - 1.0 < 0.0000001)
      printf("result correct\n");
    else
      printf("result incorrect!\n");

    return 0;
}
